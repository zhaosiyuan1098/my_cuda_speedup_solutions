#include "hip/hip_runtime.h"
#include "origin_gemm.cuh"
#include <iostream>

__global__ void origin_gemm_thread(int M, int N, int K, int *A, int *B, int *C)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N)
    {
        int sum = 0;
        for (int k = 0; k < K; k++)
        {
            sum += A[row * K + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

int* origin_gemm(args arg, int *A, int *B, int *C)
{
    dim3 threadsPerBlock(arg.M/arg.bk, arg.M/arg.bk);
    dim3 numBlocks((arg.N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (arg.M + threadsPerBlock.y - 1) / threadsPerBlock.y);
    std::cout << "numBlocks: " << numBlocks.x << " " << numBlocks.y << std::endl;
    std::cout << "threadsPerBlock: " << threadsPerBlock.x << " " << threadsPerBlock.y << std::endl;

    origin_gemm_thread<<<numBlocks, threadsPerBlock>>>(arg.M, arg.N, arg.K, A, B, C);

    hipDeviceSynchronize();
    return C;
}
