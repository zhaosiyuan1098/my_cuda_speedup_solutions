#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#include "args.h"
#include "origin_gemm.cuh"

void init_matrix(args arg, int **A, int **B, int **C)
{
    int M = arg.M;
    int K = arg.K;
    int N = arg.N;
    int bk = arg.bk;
    int rk = arg.rk;
    int grid_size = arg.grid_size;
    int block_size = arg.block_size;
    hipError_t err;
    err = hipMallocManaged(A, M * K * sizeof(int));
    if (err != hipSuccess)
    {
        std::cerr << "hipMallocManaged failed for A: " << hipGetErrorString(err) << std::endl;
        return;
    }
    err = hipMallocManaged(B, K * N * sizeof(int));
    if (err != hipSuccess)
    {
        std::cerr << "hipMallocManaged failed for B: " << hipGetErrorString(err) << std::endl;
        return;
    }
    err = hipMallocManaged(C, M * N * sizeof(int));
    if (err != hipSuccess)
    {
        std::cerr << "hipMallocManaged failed for C: " << hipGetErrorString(err) << std::endl;
        return;
    }
    for (int i = 0; i < M * K; i++)
    {
        (*A)[i] = i;
    }
    for (int i = 0; i < K * N; i++)
    {
        (*B)[i] = i;
    }
    for (int i = 0; i < M * N; i++)
    {
        (*C)[i] = 0;
    }
    std::cout << "matrix intialized!" << std::endl;
}

int main()
{
    args arg;
    int *A, *B, *C;
    init_matrix(arg, &A, &B, &C);
    int* origin_output=origin_gemm(arg, A, B, C);

    // 释放内存
    hipFree(A);
    hipFree(B);
    hipFree(C);

    return 0;
}