#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hipblas.h>

#include <stdio.h>
#include <iostream>
#include <chrono>

#include "args.h"
#include "origin_gemm.cuh"
#include "v1.cuh"
#include "v2.cuh"
#include "v3.cuh"
#include "v4.cuh"
#include "v5.cuh"
#include "v6.cuh"
#include "v7.cuh"
#include "v8.cuh"

void init_matrix(args arg, float **A, float **B, float **C)
{
    int M = arg.M;
    int K = arg.K;
    int N = arg.N;
    hipError_t err;
    err = hipMallocManaged((void **)&(*A), M * K * sizeof(float));
    if (err != hipSuccess)
    {
        std::cerr << "hipMallocManaged failed for A: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMallocManaged((void **)&(*B), K * N * sizeof(float));
    if (err != hipSuccess)
    {
        std::cerr << "hipMallocManaged failed for B: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMallocManaged((void **)&(*C), M * N * sizeof(float));
    if (err != hipSuccess)
    {
        std::cerr << "hipMallocManaged failed for C: " << hipGetErrorString(err) << std::endl;
        return;
    }
    srand(time(NULL));
    for (int i = 0; i < M * K; i++)
    {
        (*A)[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    for (int i = 0; i < K * N; i++)
    {
        (*B)[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    for (int i = 0; i < M * N; i++)
    {
        (*C)[i] = 0.0f;
    }
}

int main(int argc, char *argv[])
{
    if (argc != 3)
    {
        std::cerr << "Usage: " << argv[0] << " <MNK> <method>" << std::endl;
        return -1;
    }

    args arg;
    arg.M = std::atoi(argv[1]);
    arg.N = arg.M;
    arg.K = arg.M;
    int method = std::atoi(argv[2]);

    float *A, *B, *C, *C_cublas;
    init_matrix(arg, &A, &B, &C);
    hipMallocManaged(&C_cublas, arg.M * arg.N * sizeof(float));
    std::cout << "matrix size: " << arg.M << std::endl;
#ifdef USE_CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    const float alpha = 1.0f;
    const float beta = 0.0f;
    auto blas_start = std::chrono::high_resolution_clock::now();
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, arg.N, arg.M, arg.K, &alpha, B, arg.N, A, arg.K, &beta, C_cublas, arg.N);
    hipDeviceSynchronize();
    auto blas_end = std::chrono::high_resolution_clock::now();
    hipblasDestroy(handle);

    std::chrono::duration<float, std::milli> blas_duration = blas_end - blas_start;
    std::cout << "Method " << "cublas" << " time: " << blas_duration.count() << " ms" << std::endl;
#endif
    auto start = std::chrono::high_resolution_clock::now();

    switch (method)
    {
    case 1:
        v1(arg, A, B, C);
        break;
    case 2:
        v2(arg, A, B, C);
        break;
    case 3:
        v3(arg, A, B, C);
        break;
    case 4:
        v4(arg, A, B, C);
        break;
    case 5:
        v5(arg, A, B, C);
        break;
    case 6:
        v6(arg, A, B, C);
        break;
    case 7:
        v7(arg, A, B, C);
        break;
    case 8:
        v8(arg, A, B, C);
        break;
    default:
        std::cerr << "Invalid method!" << std::endl;
        break;
    }

    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> duration = end - start;
    std::cout << "Method " << method << " time: " << duration.count() << " ms" << std::endl;

    // 比较结果
    bool match = true;
    for (int i = 0; i < arg.M * arg.N; i++)
    {
        if (fabs(C[i] - C_cublas[i]) > 1e-3)
        {
            match = false;
            std::cout << "Results do not match at index " << i << ": " << C[i] << " != " << C_cublas[i] << std::endl;
            break;
        }
    }

    if (match)
    {
        std::cout << "Results match!" << std::endl;
    }
    else
    {
        std::cout << "Results do not match!" << std::endl;
    }
    std::cout << std::endl;

    // 释放内存
    hipFree(A);
    hipFree(B);
    hipFree(C);
    hipFree(C_cublas);

    return 0;
}