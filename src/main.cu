#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#include "args.h"
#include "origin_gemm.cuh"
#include"v1.cuh"
void init_matrix(args arg, float **A, float **B, float **C) {
    int M = arg.M;
    int K = arg.K;
    int N = arg.N;
    hipError_t err;
    err = hipMallocManaged((void**)A, M * K * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "hipMallocManaged failed for A: " << hipGetErrorString(err) << std::endl;
        return;
    }
    err = hipMallocManaged((void**)B, K * N * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "hipMallocManaged failed for B: " << hipGetErrorString(err) << std::endl;
        return;
    }
    err = hipMallocManaged((void**)C, M * N * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "hipMallocManaged failed for C: " << hipGetErrorString(err) << std::endl;
        return;
    }
    srand(time(NULL));
    for (int i = 0; i < M * K; i++) {
        (*A)[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    for (int i = 0; i < K * N; i++) {
        (*B)[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    for (int i = 0; i < M * N; i++) {
        (*C)[i] = 0.0f;
    }
    std::cout << "matrix initialized with random values!" << std::endl;
}

int main() {
    args arg;

    float *A, *B, *C;
    init_matrix(arg, &A, &B, &C);
    int* origin_output = origin_gemm(arg, reinterpret_cast<int*>(A), reinterpret_cast<int*>(B), reinterpret_cast<int*>(C));
    v1(arg, A, B, C);

    // 释放内存
    hipFree(A);
    hipFree(B);
    hipFree(C);

    return 0;
}