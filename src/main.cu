#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#include"args.h"


__global__ void origin_gemm_thread(){
    // printf can be used instead of std::cout in device code
    printf("Hello from origin_gemm!\n");
}


void init_matrix(args arg, int **A, int **B, int **C){
    int M = arg.M;
    int K = arg.K;
    int N = arg.N;
    int bk = arg.bk;
    int rk = arg.rk;
    int grid_size = arg.grid_size;
    int block_size = arg.block_size;
    hipError_t err;
    err = hipMallocManaged(A, M * K * sizeof(int));
    if (err != hipSuccess) {
        std::cerr << "hipMallocManaged failed for A: " << hipGetErrorString(err) << std::endl;
        return;
    }
    err = hipMallocManaged(B, K * N * sizeof(int));
    if (err != hipSuccess) {
        std::cerr << "hipMallocManaged failed for B: " << hipGetErrorString(err) << std::endl;
        return;
    }
    err = hipMallocManaged(C, M * N * sizeof(int));
    if (err != hipSuccess) {
        std::cerr << "hipMallocManaged failed for C: " << hipGetErrorString(err) << std::endl;
        return;
    }
    for(int i = 0; i < M * K; i++){
        (*A)[i] = i;
    }
    for(int i = 0; i < K * N; i++){
        (*B)[i] = i;
    }
    for(int i = 0; i < M * N; i++){
        (*C)[i] = 0;
    }
    std::cout<<"matrix intialized!"<<std::endl;
}

void origin_gemm(args arg ,int *A, int *B, int *C){
    origin_gemm_thread<<<1, 16>>>();
    hipDeviceSynchronize();
}

int main(){
    args arg;
    int *A, *B, *C;
    init_matrix(arg, &A, &B, &C);
    origin_gemm(arg, A, B, C);
     

    // 调用内核函数
    

    // 释放内存
    hipFree(A);
    hipFree(B);
    hipFree(C);

    return 0;
}